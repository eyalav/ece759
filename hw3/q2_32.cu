#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/types.h>
#include <sys/time.h>
#include <sys/stat.h>

#define BLOCK_DIM 32
#define SEED 26

__global__ void data(double *a, double *b, double *c, int count) {
  int t_id = blockDim.x * blockIdx.x + threadIdx.x;
  if (t_id < count)
    c[t_id] = a[t_id] + b[t_id];
}

int main (int argc, char *argv[]) {
  if (argc != 2) {
    printf("Usage: %s number\n", argv[0]);
    exit(1);
  }
  // Initialize host variables
  int i = 0;
  double *hA, *hB, *hC, *refC;
  int num = atoi(argv[1]);
  long size = num*sizeof(double);
  hA = (double *)malloc(size);
  hB = (double *)malloc(size);
  hC = (double *)malloc(size);
  refC = (double *)malloc(size);

  // Timing variables;
  struct timeval incl_start, incl_end;
  struct timeval excl_start, excl_end;
  long int incl_diff, excl_diff;

  // Populate hA, hB, refC
  srand(SEED);
  for (i=0; i<num; i++) {
    hA[i] = -10 + rand() % 20;
    hB[i] = -10 + rand() % 20;
    refC[i] = hA[i] + hB[i];
  }

  // Device memory allocation
  double *dA, *dB, *dC;
  hipMalloc((void **)&dA, size);
  hipMalloc((void **)&dB, size);
  hipMalloc((void **)&dC, size);
  gettimeofday(&incl_start, NULL);
  hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
  hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);
  
  // Launch kernel
  int GRID_DIM = (num % BLOCK_DIM == 0) 
                ? num / BLOCK_DIM 
                : (int)(num / BLOCK_DIM) + 1;
  dim3 dimGrid(GRID_DIM, 1, 1);
  dim3 dimBlock(BLOCK_DIM, 1, 1);
  gettimeofday(&excl_start, NULL);
  data<<<dimGrid, dimBlock>>>(dA, dB, dC, num);
  gettimeofday(&excl_end, NULL);
  hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);
  gettimeofday(&incl_end, NULL);

  // Verify results
  for (i=0; i<num; i++) {
    if (fabs(refC[i] - hC[i]) > 1e-12) {
      printf("FAIL\n");
      exit(1);
    }
  }

  // Print metrics
  incl_diff = (incl_end.tv_usec + 1000000 * incl_end.tv_sec) - (incl_start.tv_usec + 1000000 * incl_start.tv_sec);
  excl_diff = (excl_end.tv_usec + 1000000 * excl_end.tv_sec) - (excl_start.tv_usec + 1000000 * excl_start.tv_sec);
  //printf("Inclusive: %ld\n", incl_diff);
  //printf("Exclusive: %ld\n", excl_diff);
  printf("Size = %d\n Inclusive Time = %ld\n Exclusive Time = %ld\n", num, incl_diff, excl_diff);

  // Cleanup
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
  free(hA);
  free(hB);
  free(hC);
  free(refC);

  return 0;
}

