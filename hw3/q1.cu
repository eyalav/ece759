#include <stdio.h>
#include <hip/hip_runtime.h>

#define NUM 16

__global__ void data(int *array) {
  int t_id = blockDim.x * blockIdx.x + threadIdx.x;
  array[t_id] = threadIdx.x + blockIdx.x;
  printf("d_array[%d] = %d\n", t_id, array[t_id]);
}

int main () {
  // Initialize variables
  int h_array[NUM];
  int *d_array;
  size_t size = NUM*sizeof(int);
  int i = 0;

  // Device memory allocation
  dim3 dimGrid(2, 1, 1);
  dim3 dimBlock(8, 1, 1);
  hipMalloc((void **)&d_array, size);
  printf("Device memory:\n");
  data<<<dimGrid, dimBlock>>>(d_array);
  hipMemcpy(h_array, d_array, size, hipMemcpyDeviceToHost);

  // Print data copied to host
  printf("Host memory:\n");
  for (i = 0; i < NUM; i++){
    printf("h_array[%d] = %d\n", i, h_array[i]);
  }

  // Cleanup
  hipFree(d_array);

  return 0;
}

