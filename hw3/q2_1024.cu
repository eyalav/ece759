#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/types.h>
#include <sys/time.h>
#include <sys/stat.h>

#define BLOCK_DIM 1024
#define SEED 26

__global__ void data(double *a, double *b, double *c, int count) {
  int t_id = blockDim.x * blockIdx.x + threadIdx.x;
  if (t_id < count)
    c[t_id] = a[t_id] + b[t_id];
}

int main (int argc, char *argv[]) {
  if (argc != 2) {
    printf("Usage: %s number\n", argv[0]);
    exit(1);
  }
  // Initialize host variables
  int i = 0;
  double *hA, *hB, *hC, *refC;
  int num = atoi(argv[1]);
  long size = num*sizeof(double);
  hA = (double *)malloc(size);
  hB = (double *)malloc(size);
  hC = (double *)malloc(size);
  refC = (double *)malloc(size);

  // Timing variables;
  hipEvent_t incl_start, incl_end;
  hipEvent_t excl_start, excl_end;
  float time_incl, time_excl;
  hipEventCreate(&incl_start);
  hipEventCreate(&incl_end);
  hipEventCreate(&excl_start);
  hipEventCreate(&excl_end);

  // Populate hA, hB, refC
  srand(SEED);
  for (i=0; i<num; i++) {
    hA[i] = -10 + rand() % 20;
    hB[i] = -10 + rand() % 20;
    refC[i] = hA[i] + hB[i];
  }

  // Device memory allocation
  double *dA, *dB, *dC;
  hipMalloc((void **)&dA, size);
  hipMalloc((void **)&dB, size);
  hipMalloc((void **)&dC, size);
  hipEventRecord(incl_start, 0);
  hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
  hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);
  
  // Launch kernel
  int GRID_DIM = (num % BLOCK_DIM == 0) 
                ? num / BLOCK_DIM 
                : (int)(num / BLOCK_DIM) + 1;
  dim3 dimGrid(GRID_DIM, 1, 1);
  dim3 dimBlock(BLOCK_DIM, 1, 1);
  hipEventRecord(excl_start, 0);
  data<<<dimGrid, dimBlock>>>(dA, dB, dC, num);
  hipEventRecord(excl_end, 0);
  hipEventSynchronize(excl_end);
  hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);
  hipEventRecord(incl_end, 0);
  hipEventSynchronize(incl_end);

  // Verify results
  for (i=0; i<num; i++) {
    if (fabs(refC[i] - hC[i]) > 1e-12) {
      printf("FAIL\n");
      exit(1);
    }
  }

  // Print metrics
  hipEventElapsedTime(&time_incl, incl_start, incl_end);
  hipEventElapsedTime(&time_excl, excl_start, excl_end);
  //time_incl *= 1000;
  //time_excl *= 1000;
  //printf("Inclusive: %f\n", incl_diff);
  //printf("Exclusive: %f\n", excl_diff);
  //printf("Size = %d\n Inclusive Time = %f\n Exclusive Time = %f\n", num, time_incl, time_excl);
  printf("%d\t%f\t%f\n", num, time_incl, time_excl);

  // Cleanup
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
  free(hA);
  free(hB);
  free(hC);
  free(refC);

  return 0;
}

