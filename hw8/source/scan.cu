#include "hip/hip_runtime.h"
#include<iostream>
#include<stdlib.h>
#include<stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <assert.h>
#include "scan_gold.cpp"
#include "kernel.cu"

#define BLOCK_SIZE 1024 

int checkResults(int startElem, int endElem, float* cudaRes, float* res)
{
    int nDiffs=0;
    const float smallVal = 0.000001f;
    for(int i=startElem; i<endElem; i++)
        if(fabs(cudaRes[i]-res[i])>smallVal)
            nDiffs++;
    return nDiffs;
}

void initializeArray(float* arr, int nElements)
{
    const int myMinNumber = -5;
    const int myMaxNumber = 5;
    srand(26);
    for( int i=0; i<nElements; i++)
      arr[i] = (float)(rand() % (myMaxNumber - myMinNumber + 1) + myMinNumber);
      //arr[i] = 1.f;
}

int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("Missing input N\n");                                                               
    exit(1);                                                                                   
    }
  int N = atoi(argv[1]);
  int size = N * sizeof(float); 
  
  //allocate resources
  float *h_in;
  //h_in = (float *)malloc(size);
  //assert(h_in != NULL);
  assert(hipSuccess == hipHostMalloc(&h_in, size));
  
  float *h_out;
  //h_out = (float *)malloc(size); 
  //assert(h_out != NULL);
  assert(hipSuccess == hipHostMalloc(&h_out, size));
  
  float *cuda_out;
  //cuda_out = (float *)malloc(size); 
  //assert(cuda_out != NULL);
  assert(hipSuccess == hipHostMalloc(&cuda_out, size));
  
  float *d_in;      
  assert(hipSuccess == hipMalloc(&d_in, size));
  
  float *d_out;     
  assert(hipSuccess == hipMalloc(&d_out, size));

  // Initialize input array
  initializeArray(h_in, N);

  // Timing variables
  float cpu, gpu;
  hipEvent_t gpu_start, gpu_end;
  hipEvent_t cpu_start, cpu_end;
  hipEventCreate(&gpu_start);
  hipEventCreate(&gpu_end);
  hipEventCreate(&cpu_start);
  hipEventCreate(&cpu_end);
  
  dim3 grid_simple, grid_tree;
  grid_simple.x = (N + BLOCK_SIZE - 1)/(BLOCK_SIZE);
  grid_tree.x = (N + 2*BLOCK_SIZE - 1)/(2*BLOCK_SIZE);
  dim3 block(BLOCK_SIZE, 1, 1);
  int blocks_left = grid_simple.x - 1;
  
  hipEventRecord(gpu_start, NULL);

  assert(hipSuccess == hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice));
  //hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);
  
  scan_simple<<<grid_simple, block, sizeof(float)*BLOCK_SIZE*2>>>(d_out, d_in, N);
  //scan_tree<<<grid_tree, block, sizeof(float)*BLOCK_SIZE*2>>>(d_out, d_in, N);
  //int blocks_left = grid_tree.x - 1;

  // Cannot process more than 8192 block in one go
  // Max. shared limit of 48k
  if (blocks_left > 0)
    reduce<<<blocks_left, BLOCK_SIZE, sizeof(float)*(BLOCK_SIZE)>>>(d_out, d_in);
    //reduce<<<blocks_left, BLOCK_SIZE*2, sizeof(float)*(BLOCK_SIZE*2)>>>(d_out);
    //reduce2<<<grid_simple, block>>>(d_out, d_in, BLOCK_SIZE, N);

  
  assert(hipSuccess == hipMemcpy(cuda_out, d_out, size, hipMemcpyDeviceToHost));
  //hipMemcpy(cuda_out, d_out, size, hipMemcpyDeviceToHost);
  
  hipEventRecord(gpu_end, NULL);
  hipEventSynchronize(gpu_end);
  hipEventElapsedTime(&gpu, gpu_start, gpu_end);

  hipEventRecord(cpu_start, NULL);
 
  // Compute on CPU
  computeGold(h_out, h_in, N);
  
  hipEventRecord(cpu_end, NULL);
  hipEventSynchronize(cpu_end);
  hipEventElapsedTime(&cpu, cpu_start, cpu_end);

  int nDiffs = checkResults(1, N-1, cuda_out, h_out + 1);
  std::cout << "Size = " << N << "\n";
  nDiffs==0? std::cout<<"Looks good.\n": std::cout<<"Doesn't look good: " << nDiffs << " differences\n";
  std::cout << "GPU time =  " << gpu << "\n";
  std::cout << "CPU time =  " << cpu << "\n";
  std::cout << "Speedup = " << cpu/gpu << "\n";

  //free resources
  hipFree(h_in); 
  hipFree(h_out); 
  hipFree(cuda_out);
  hipFree(d_in);  
  hipFree(d_out);
  return 0;
}
