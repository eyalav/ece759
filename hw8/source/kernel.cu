
#include <hip/hip_runtime.h>

__global__ void scan_simple(float *out, float *in, int length) {
  volatile extern __shared__ float data[];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int tx = threadIdx.x;
  data[tx] = in[tid];
  int pout = 0; int pin = 1;
  
  if (tid < length) {
  for (int offset = 1; offset < blockDim.x; offset <<= 1) {
    pout = 1 - pout;
    pin = 1 - pin;

    if (tx >= offset)
      data[pout * blockDim.x + tx] = data[pin * blockDim.x + tx] + data[pin * blockDim.x + tx - offset];
    else 
      data[pout * blockDim.x + tx] = data[pin * blockDim.x + tx];

    __syncthreads();
  }

  out[tid] = data[pout * blockDim.x + tx];
  if (tx == blockDim.x - 1) in[tid] = data[pout * blockDim.x + tx];
  }
}

__global__ void scan_tree(float *out, float *in, int length) {
  volatile extern __shared__ float data[];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int tx = threadIdx.x;
  int offset = 1;
  data[2*tx] = in[2*tid];
  data[2*tx + 1] = in[2*tid + 1];

  // Sum up
  for (int jump = blockDim.x; jump > 0; jump >>= 1) {
    __syncthreads();
    if (tx < jump) {
      int a = offset * (2*tx + 1) - 1;
      int b = offset * (2*tx + 2) - 1;
      data[b] += data[a];
    }
    offset <<= 1;
  }

  if (tx == 0) data[2*blockDim.x - 1] = in[2*blockDim.x * (blockIdx.x + 1) - 1];

  // Sum down
  for (int jump = 1; jump < blockDim.x<<1; jump <<= 1) {
    offset >>= 1;
    __syncthreads();

    if (tx < jump) {
      int a = offset * (2*tx + 1) - 1;
      int b = offset * (2*tx + 2) - 1;

      float temp = data[a];
      data[a] = data[b];
      data[b] += temp;
    }
  }
  __syncthreads();
  if (tx == 0) data[0] = in[tid];

  out[2*tid] = data[2*tx];
  out[2*tid + 1] = data[2*tx + 1];
}

__global__ void reduce(float *g_data, float *old_data) {
  volatile extern __shared__ float data[];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int bid = blockIdx.x + 1;
  int tx = threadIdx.x;

  // Load partially scanned blocks and partial sums
  data[tx] = g_data[tid + blockDim.x];
  
  // Add partial sum
  for (int i = 0; i < bid; i++)
    data[tx] += old_data[blockDim.x + i*blockDim.x - 1];
  __syncthreads();

  // Write back to global memory
  g_data[tid + blockDim.x] = data[tx];
}
