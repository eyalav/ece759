#include "hip/hip_runtime.h"
#include<iostream>
#include<stdlib.h>
#include<stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <assert.h>
#include "scan_gold.cpp"

#define BLOCK_SIZE 1024

int checkResults(int startElem, int endElem, float* cudaRes, float* res)
{
    int nDiffs=0;
    const float smallVal = 0.000001f;
    for(int i=startElem; i<endElem; i++)
        if(fabs(cudaRes[i]-res[i])>smallVal)
            nDiffs++;
    return nDiffs;
}

void initializeArray(float* arr, int nElements)
{
    const int myMinNumber = -5;
    const int myMaxNumber = 5;
    srand(26);
    for( int i=0; i<nElements; i++)
      //arr[i] = (float)(rand() % (myMaxNumber - myMinNumber + 1) + myMinNumber);
      arr[i] = 1.f;
}

__global__ void scan_simple(float *out, float *in, int length) {
  extern __shared__ float data[];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int tx = threadIdx.x;
  data[tx] = in[tid];
  int pout = 0; int pin = 1;
  
  if (tid < length) {
  for (int offset = 1; offset < blockDim.x; offset <<= 1) {
    pout = 1 - pout;
    pin = 1 - pin;

    if (tx >= offset)
      data[pout * blockDim.x + tx] = data[pin * blockDim.x + tx] + data[pin * blockDim.x + tx - offset];
    else 
      data[pout * blockDim.x + tx] = data[pin * blockDim.x + tx];

    __syncthreads();
  }

  out[tid] = data[pout * blockDim.x + tx];
  }
}

__global__ void scan_tree(float *out, float *in, int length) {
  extern __shared__ float data[];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int tx = threadIdx.x;
  int offset = 1;
  data[2*tx] = in[2*tid];
  data[2*tx + 1] = in[2*tid + 1];

  // Sum up
  for (int jump = blockDim.x; jump > 0; jump >>= 1) {
    __syncthreads();
    if (tx < jump) {
      int a = offset * (2*tx + 1) - 1;
      int b = offset * (2*tx + 2) - 1;
      data[b] += data[a];
    }
    offset <<= 1;
  }

  if (tx == 0) data[length - 1] = in[tid];

  // Sum down
  for (int jump = 1; jump < blockDim.x<<1; jump <<= 1) {
    offset >>= 1;
    __syncthreads();

    if (tx < jump) {
      int a = offset * (2*tx + 1) - 1;
      int b = offset * (2*tx + 2) - 1;

      float temp = data[a];
      data[a] = data[b];
      data[b] += temp;
    }
  }
  __syncthreads();

  out[2*tid] = data[2*tx];
  out[2*tid + 1] = data[2*tx + 1];
}

int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("Missing input N\n");                                                               
    exit(1);                                                                                   
    }
  int N = atoi(argv[1]);
  int size = N * sizeof(float); 
  
  //allocate resources
  float *h_in      = (float *)malloc(size);
  assert(h_in != NULL);
  
  float *h_out     = (float *)malloc(size); 
  assert(h_out != NULL);
  
  float *cuda_out  = (float *)malloc(size); 
  assert(cuda_out != NULL);
  
  float *d_in;      
  assert(hipSuccess == hipMalloc(&d_in, size));
  
  float *d_out;     
  assert(hipSuccess == hipMalloc(&d_out, size));

  // Initialize input array
  initializeArray(h_in, N);

  // Timing variables
  float cpu, gpu;
  hipEvent_t gpu_start, gpu_end;
  hipEvent_t cpu_start, cpu_end;
  hipEventCreate(&gpu_start);
  hipEventCreate(&gpu_end);
  hipEventCreate(&cpu_start);
  hipEventCreate(&cpu_end);
  
  dim3 grid_simple, grid_tree;
  grid_simple.x = (N + BLOCK_SIZE - 1)/(BLOCK_SIZE);
  grid_tree.x = (N + 2*BLOCK_SIZE - 1)/(2*BLOCK_SIZE);
  dim3 block(BLOCK_SIZE, 1, 1);
  
  hipEventRecord(gpu_start, NULL);

  assert(hipSuccess == hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice));
  
  //scan_simple<<<grid_simple, block, sizeof(float)*BLOCK_SIZE*2>>>(d_out, d_in, N);
  scan_tree<<<grid_tree, block, sizeof(float)*BLOCK_SIZE*2>>>(d_out, d_in, N);
  
  assert(hipSuccess == hipMemcpy(cuda_out, d_out, size, hipMemcpyDeviceToHost));
  
  hipEventRecord(gpu_end, NULL);
  hipEventSynchronize(gpu_end);
  hipEventElapsedTime(&gpu, gpu_start, gpu_end);

  hipEventRecord(cpu_start, NULL);
 
  // Compute on CPU
  computeGold(h_out, h_in, N);
  
  hipEventRecord(cpu_end, NULL);
  hipEventSynchronize(cpu_end);
  hipEventElapsedTime(&cpu, cpu_start, cpu_end);

  int nDiffs = checkResults(1, N-1, cuda_out, h_out + 1);
  nDiffs==0? std::cout<<"Looks good.\n": std::cout<<"Doesn't look good: " << nDiffs << "differences\n";
  std::cout << "GPU time =  " << gpu << "\n";
  std::cout << "CPU time =  " << cpu << "\n";

  //free resources
  free(h_in); 
  free(h_out); 
  free(cuda_out);
  hipFree(d_in);  
  hipFree(d_out);
  return 0;
}
