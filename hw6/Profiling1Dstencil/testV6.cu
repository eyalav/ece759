#include<iostream>
#include<stdlib.h>
#include<stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

//#define N 1000000
#define RADIUS 3

int checkResults(int startElem, int endElem, float* cudaRes, float* res)
{
    int nDiffs=0;
    const float smallVal = 0.000001f;
    for(int i=startElem; i<endElem; i++)
        if(fabs(cudaRes[i]-res[i])>smallVal)
            nDiffs++;
    return nDiffs;
}

void initializeWeights(float* weights, int rad)
{
    // for now hardcoded for RADIUS=3
    weights[0] = 0.50f;
    weights[1] = 0.75f;
    weights[2] = 1.25f;
    weights[3] = 2.00f;
    weights[4] = 1.25f;
    weights[5] = 0.75f;
    weights[6] = 0.50f;
}

void initializeArray(float* arr, int nElements)
{
    const int myMinNumber = -5;
    const int myMaxNumber = 5;
    srand(time(NULL));
    for( int i=0; i<nElements; i++)
        arr[i] = (float)(rand() % (myMaxNumber - myMinNumber + 1) + myMinNumber);
}

void applyStencil1D_SEQ(int sIdx, int eIdx, const float *weights, float *in, float *out) {
  
  for (int i = sIdx; i < eIdx; i++) {   
    out[i] = 0;
    //loop over all elements in the stencil
    for (int j = -RADIUS; j <= RADIUS; j++) {
      out[i] += weights[j + RADIUS] * in[i + j]; 
    }
    out[i] = out[i] / (2 * RADIUS + 1);
  }
}

__global__ void applyStencil1D(int sIdx, int eIdx, const float *weights, const float *input, float *out) {
    __shared__ float in[RADIUS * 2 + 1027];
    __shared__ float sw[RADIUS * 2 + 1];
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int x = threadIdx.x + RADIUS;
    
    in[x] = input[tid];
    if (threadIdx.x < RADIUS) {
      in[x - RADIUS] = input[tid - RADIUS];
      in[x + blockDim.x] = input[tid + blockDim.x];
    }
    if (threadIdx.x < RADIUS * 2 + 1)
      sw[threadIdx.x] = weights[threadIdx.x];
    __syncthreads();
    
    float result = 0.f;
    if (tid < eIdx) {
    result += sw[0]*in[x-3];
    result += sw[1]*in[x-2];
    result += sw[2]*in[x-1];
    result += sw[3]*in[x];
    result += sw[4]*in[x+1];
    result += sw[5]*in[x+2];
    result += sw[6]*in[x+3];
    result /=7.f;
    }
    out[tid] = result;
}

int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("Missing input N\n");
    exit(1);
  }
  int N = atoi(argv[1]);
  int size = N * sizeof(float); 
  int wsize = (2 * RADIUS + 1) * sizeof(float); 
  //allocate resources
  float *weights   ; hipHostMalloc(&weights, wsize, hipHostMallocDefault);
  float *in        ; hipHostMalloc(&in, size, hipHostMallocDefault);
  float *out       ; hipHostMalloc(&out, size, hipHostMallocDefault);
  float *cuda_out  ; hipHostMalloc(&cuda_out, size, hipHostMallocDefault);
  initializeWeights(weights, RADIUS);
  initializeArray(in, N);
  float *d_weights;  hipMalloc((void **)&d_weights, wsize);
  float *d_in;       hipMalloc((void **)&d_in, size);
  float *d_out;      hipMalloc((void **)&d_out, size);
  
  // Timing variables
  float cpu, gpu;
  hipEvent_t gpu_start, gpu_end;
  hipEvent_t cpu_start, cpu_end;
  hipEventCreate(&gpu_start);
  hipEventCreate(&gpu_end);
  hipEventCreate(&cpu_start);
  hipEventCreate(&cpu_end);

  hipEventRecord(gpu_start, NULL);
  hipMemcpy(d_weights,weights,wsize,hipMemcpyHostToDevice);
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
  applyStencil1D<<<((N+1023)/1024), 1024>>>(RADIUS, N-RADIUS, d_weights, d_in, d_out);
  hipMemcpy(cuda_out, d_out, size, hipMemcpyDeviceToHost);
  hipEventRecord(gpu_end, NULL);
  hipEventSynchronize(gpu_end);
  hipEventElapsedTime(&gpu, gpu_start, gpu_end);
  
  hipEventRecord(cpu_start, NULL);
  applyStencil1D_SEQ(RADIUS, N-RADIUS, weights, in, out);
  hipEventRecord(cpu_end, NULL);
  hipEventSynchronize(cpu_end);
  hipEventElapsedTime(&cpu, cpu_start, cpu_end);

  int nDiffs = checkResults(RADIUS, N-RADIUS, cuda_out, out);
  nDiffs==0? std::cout<<"Looks good.\n": std::cout<<"Doesn't look good: " << nDiffs << "differences\n";
  std::cout << "GPU time =  " << gpu*1000 << "\n";
  std::cout << "CPU time =  " << cpu*1000 << "\n";

  //free resources
  hipFree(weights); hipFree(in); hipFree(out); hipFree(cuda_out);
  hipFree(d_weights);  hipFree(d_in);  hipFree(d_out);
  return 0;
}
