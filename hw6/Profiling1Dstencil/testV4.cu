#include<iostream>
#include<stdlib.h>
#include<stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

//#define N 1000000
#define RADIUS 3

int checkResults(int startElem, int endElem, float* cudaRes, float* res)
{
    int nDiffs=0;
    const float smallVal = 0.000001f;
    for(int i=startElem; i<endElem; i++)
        if(fabs(cudaRes[i]-res[i])>smallVal)
            nDiffs++;
    return nDiffs;
}

void initializeWeights(float* weights, int rad)
{
    // for now hardcoded for RADIUS=3
    weights[0] = 0.50f;
    weights[1] = 0.75f;
    weights[2] = 1.25f;
    weights[3] = 2.00f;
    weights[4] = 1.25f;
    weights[5] = 0.75f;
    weights[6] = 0.50f;
}

void initializeArray(float* arr, int nElements)
{
    const int myMinNumber = -5;
    const int myMaxNumber = 5;
    srand(time(NULL));
    for( int i=0; i<nElements; i++)
        arr[i] = (float)(rand() % (myMaxNumber - myMinNumber + 1) + myMinNumber);
}

void applyStencil1D_SEQ(int sIdx, int eIdx, const float *weights, float *in, float *out) {
  
  for (int i = sIdx; i < eIdx; i++) {   
    out[i] = 0;
    //loop over all elements in the stencil
    for (int j = -RADIUS; j <= RADIUS; j++) {
      out[i] += weights[j + RADIUS] * in[i + j]; 
    }
    out[i] = out[i] / (2 * RADIUS + 1);
  }
}

__global__ void applyStencil1D(int sIdx, int eIdx, const float *weights, float *in, float *out) {
    int i = sIdx + blockIdx.x*blockDim.x + threadIdx.x;
    if( i < eIdx ) {
        float result = 0.f;
        result += weights[0]*in[i-3];
        result += weights[1]*in[i-2];
        result += weights[2]*in[i-1];
        result += weights[3]*in[i];
        result += weights[4]*in[i+1];
        result += weights[5]*in[i+2];
        result += weights[6]*in[i+3];
        result /=7.f;
        out[i] = result;
    }
}

int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("Missing input N\n");                                                               
    exit(1);                                                                                   
    }
  int N = atoi(argv[1]);
  int size = N * sizeof(float); 
  int wsize = (2 * RADIUS + 1) * sizeof(float); 
  //allocate resources
  float *weights = (float *)malloc(wsize);
  float *in      = (float *)malloc(size);
  float *out     = (float *)malloc(size); 
  float *cuda_out= (float *)malloc(size); 
  initializeWeights(weights, RADIUS);
  initializeArray(in, N);
  float *d_weights;  hipMalloc(&d_weights, wsize);
  float *d_in;       hipMalloc(&d_in, size);
  float *d_out;      hipMalloc(&d_out, size);

  // Timing variables
  float cpu, gpu;
  hipEvent_t gpu_start, gpu_end;
  hipEvent_t cpu_start, cpu_end;
  hipEventCreate(&gpu_start);
  hipEventCreate(&gpu_end);
  hipEventCreate(&cpu_start);
  hipEventCreate(&cpu_end);
  
  hipEventRecord(gpu_start, NULL);
  hipMemcpy(d_weights,weights,wsize,hipMemcpyHostToDevice);
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
 
  dim3 grid;
  if (N < 67108864)
    grid.x = (N+1023)/1024;
  else {
    grid.x = (N+1023)/1024/2;
    grid.y = 2;
  }
  dim3 block(1024, 1, 1);
  
  applyStencil1D<<<grid, block>>>(RADIUS, N-RADIUS, d_weights, d_in, d_out);
  hipMemcpy(cuda_out, d_out, size, hipMemcpyDeviceToHost);
  hipEventRecord(gpu_end, NULL);
  hipEventSynchronize(gpu_end);
  hipEventElapsedTime(&gpu, gpu_start, gpu_end);

  hipEventRecord(cpu_start, NULL);
  applyStencil1D_SEQ(RADIUS, N-RADIUS, weights, in, out);
  hipEventRecord(cpu_end, NULL);
  hipEventSynchronize(cpu_end);
  hipEventElapsedTime(&cpu, cpu_start, cpu_end);

  int nDiffs = checkResults(RADIUS, N-RADIUS, cuda_out, out);
  nDiffs==0? std::cout<<"Looks good.\n": std::cout<<"Doesn't look good: " << nDiffs << "differences\n";
  std::cout << "GPU time =  " << gpu*1000 << "\n";
  std::cout << "CPU time =  " << cpu*1000 << "\n";

  //free resources
  free(weights); free(in); free(out); free(cuda_out);
  hipFree(d_weights);  hipFree(d_in);  hipFree(d_out);
  return 0;
}
