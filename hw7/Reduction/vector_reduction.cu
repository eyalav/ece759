#include "hip/hip_runtime.h"
/*
* Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
*
* NOTICE TO USER:   
*
* This source code is subject to NVIDIA ownership rights under U.S. and 
* international Copyright laws.  
*
* This software and the information contained herein is PROPRIETARY and 
* CONFIDENTIAL to NVIDIA and is being provided under the terms and 
* conditions of a Non-Disclosure Agreement.  Any reproduction or 
* disclosure to any third party without the express written consent of 
* NVIDIA is prohibited.     
*
* NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
* CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
* IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
* REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
* MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
* IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
* OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
* OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
* OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
* OR PERFORMANCE OF THIS SOURCE CODE.  
*
* U.S. Government End Users.  This source code is a "commercial item" as 
* that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
* "commercial computer software" and "commercial computer software 
* documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
* and is provided to the U.S. Government only as a commercial end item.  
* Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
* 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
* source code with only those rights set forth herein.
*/

#ifdef _WIN32
#  define NOMINMAX 
#endif

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <fstream>
#include <omp.h>
using namespace std;
// includes, project

// includes, kernels
#include "vector_reduction_kernel.h"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

double computeOnDevice(double* h_data, int array_mem_size);

extern "C" void computeGold( double* reference, double* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int 
main( int argc, char** argv) 
{
    runTest( argc, argv);
    return EXIT_SUCCESS;
}

////////////////////////////////////////////////////////////////////////////////
//! Run test
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    unsigned int num_elements;
    int M = 0;

    if (argc != 3) {
      printf ("usgae: %s number M\n", argv[0]);
      exit(1);
    }

    num_elements = atoi(argv[1]);
    M = atoi(argv[2]);
    
    const unsigned int array_mem_size = sizeof(double) * num_elements;
    double* h_data;
    if ((h_data = (double*) malloc(array_mem_size)) == NULL)
      perror("malloc");
    //if (posix_memalign((void **)&h_data, 8, array_mem_size) != 0)
     // perror("aligned malloc");
    srand(26);
    for( unsigned int i = 0; i < num_elements; ++i) 
      h_data[i] = (double)((1000*rand()) % (2 * M + 1) - M);

    // compute reference solution
    double reference = 0.0f;  
    float cpu;
    hipEvent_t cpu_start, cpu_end;
    hipEventCreate(&cpu_start);
    hipEventCreate(&cpu_end);
    hipEventRecord(cpu_start, NULL);
    computeGold(&reference , h_data, num_elements);
    hipEventRecord(cpu_end, NULL);
    hipEventSynchronize(cpu_end);
    hipEventElapsedTime(&cpu, cpu_start, cpu_end);
    printf("Cpu time = %f\n", cpu);
    
    // **===-------- Modify the body of this function -----------===**
    double result = computeOnDevice(h_data, num_elements);
    // **===-----------------------------------------------------------===**


    // Run accuracy test
    double epsilon = 0.0001;
    unsigned int result_regtest = (abs(result - reference) <= epsilon);
    printf( "Test %s\n", (1 == result_regtest) ? "PASSED" : "FAILED");
    printf( "device: %lf  host: %lf\n", result, reference);
    // cleanup memory
    free( h_data);
}

// **===----------------- Modify this function ---------------------===**
// Take h_data from host, copies it to device, setup grid and thread 
// dimensions, excutes kernel function, and copy result of scan back
// to h_data.
// Note: double* h_data is both the input and the output of this function.
double computeOnDevice(double* h_data, int num_elements)
{
  double result;
  double *d_data;
  int memSize = sizeof(double)*num_elements;
  hipError_t err;
  // Allocate device memory
  err = hipMalloc((void **)&d_data, memSize);
  if (err != hipSuccess)
    printf("Error in hipMalloc\n");
    
  float gpu;
  hipEvent_t gpu_start, gpu_end;
  hipEventCreate(&gpu_start);
  hipEventCreate(&gpu_end);

  // Copy data to device
  hipEventRecord(gpu_start, NULL);
  err = hipMemcpy(d_data, h_data, memSize, hipMemcpyHostToDevice);
  //if (err != hipSuccess)
  //  printf("Error in hipMemcpy\n");

  // Execute kernel
  int block = 1024;
  unsigned int num_blocks = (num_elements+1023)/1024;
  reduction<<<num_blocks, block>>>(d_data, num_elements);
  do {
  int new_elements = num_blocks;
  num_blocks = (new_elements+1023)/1024;
  //printf("num_blocks = %d\n", num_blocks);
  reduction<<<num_blocks, block>>>(d_data, new_elements);
  } while (num_blocks > 1);

  // Get data back
  err = hipMemcpy(&result, d_data, sizeof(double), hipMemcpyDeviceToHost);
  hipEventRecord(gpu_end, NULL);
  hipEventSynchronize(gpu_end);
  hipEventElapsedTime(&gpu, gpu_start, gpu_end);
  printf("Gpu time = %f\n", gpu);
  //if (err != hipSuccess)
  //  printf("Error in hipMemcpy\n");

  // Cleanup and return the result
  hipFree(d_data);
  return result;

}
     
