#include <stdio.h>
#include <hip/hip_runtime.h>

int main (int argc, char *argv[]) {
  // Initialize variables
  if (argc != 2)
    exit(1);
  size_t size = atoi(argv[1]);

  void *host, *host2, *device, *device2;

  host = malloc(size);
  if (host == NULL)
    perror("malloc");

  if (hipSuccess != hipHostMalloc(&host2, size, hipHostMallocDefault))
    perror("hipHostMalloc");

  if (hipSuccess != hipMalloc(&device, size))
    perror("hipMalloc");

  hipEvent_t start, end;
  float time;
  hipEventCreate(&start);
  hipEventCreate(&end);

  hipEventRecord(start, 0);
  hipMemcpy(device, host, size, hipMemcpyHostToDevice);
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  hipEventElapsedTime(&time, start, end);
  printf ("%f\t", time);

  hipEventRecord(start, 0);
  hipMemcpy(device, host2, size, hipMemcpyHostToDevice);
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  hipEventElapsedTime(&time, start, end);
  printf ("%f\t", time);
  hipFree(device);

  if (hipSuccess != hipMalloc(&device2, size))
    perror("hipMalloc");

  hipEventRecord(start, 0);
  hipMemcpy(host, device2, size, hipMemcpyDeviceToHost);
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  hipEventElapsedTime(&time, start, end);
  printf ("%f\t", time);
 
  hipEventRecord(start, 0);
  hipMemcpy(host2, device2, size, hipMemcpyDeviceToHost);
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  hipEventElapsedTime(&time, start, end);
  printf ("%f\n", time);
  
  // Cleanup
  free(host);
  hipFree(host2);
  hipFree(device2);

  return 0;
}

