#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define BLOCK_DIM 16

__global__ void multiply(int *a, int *b, int *c, int wa) {
  int t_id = threadIdx.x;
  int i;
  c[t_id] = 0;
  for (i=0; i<wa; i++)
    c[t_id] += a[t_id * wa + i] * b[i];
}

int main (int argc, char *argv[]) {
  // Initialize host variables
  int i, j;
  int A[16][32];
  int b[32];
  int c[16];
  for (i=0; i<16; i++)
      for (j=0; j<32; j++)
        A[i][j] = i + j;
  for (i=0; i<32; i++)
    b[i] = i;

  // Timing variables;
  hipEvent_t start, end;
  float time;
  hipEventCreate(&start);
  hipEventCreate(&end);

  // Device memory allocation
  int *dA, *dB, *dC;
  hipMalloc((void **)&dA, sizeof(int)*16*32);
  hipMalloc((void **)&dB, sizeof(int)*32);
  hipMalloc((void **)&dC, sizeof(int)*16);
  hipEventRecord(start, 0);
  hipMemcpy(dA, A, sizeof(int)*16*32, hipMemcpyHostToDevice);
  hipMemcpy(dB, b, sizeof(int)*32, hipMemcpyHostToDevice);
  
  // Launch kernel
  multiply<<<1, 16>>>(dA, dB, dC, 32);
  hipMemcpy(c, dC, sizeof(int)*16, hipMemcpyDeviceToHost);
  hipEventRecord(end, 0);
  hipEventSynchronize(end);

  hipEventElapsedTime(&time, start, end);

  FILE *out;
  out = fopen("Problem1.out", "w");
  for (i=0; i<16; i++)
    fprintf(out, "%d\n", c[i]);

  hipDeviceProp_t deviceProp;
  const int currentDevice = 0;
  if (hipGetDeviceProperties(&deviceProp, currentDevice) == hipSuccess) 
    printf("Device %d: %s\n", currentDevice, deviceProp.name);
  printf("Inclusive time = %fms\n", time);
  printf("Result in the file Problem1.out\n");

  // Cleanup
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);

  return 0;
}

